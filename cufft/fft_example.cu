#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <complex>
#include <vector>
#include <iostream>
#include "../common/timers.hpp"
#include "../common/error_macros.h"
#define N_DIM 8192
#define N_PRINTS 30



namespace dft_contants 
{
    const std::complex<double> PI {{ 3.14159, 0 }};
    const std::complex<double> IM_UNIT {{0, 1}};

    const double FREQ = 3.0;

}
namespace dft_functions 
{
    std::vector<std::complex<double>> dft(const std::vector<std::complex<double> > &x) 
    {
        std::vector<std::complex<double> > dft(x.size()); // output dft
        double xN = double(x.size()); // samples 

        std::complex<double> temp = {{0, 0 }};  // temp accumulator

        for (int k = 0; k < x.size(); ++k) {
            for (int n = 0; n < x.size(); ++n) {
                temp = {{double(-1*2*k*n) / xN, 0 }};
                dft[k] += x[n] * exp(dft_contants::IM_UNIT * dft_contants::PI * temp); // divide by N to normalize
            }
        }
        return dft; 
    }

    void generate_fake_signal(std::vector<std::complex<double> > &x, size_t dim) 
    {
        double delta = (dft_contants::PI.real() / dft_contants::FREQ);
        for (int i = 0; i < dim; ++i) {
            x.push_back(cos(i * delta));
        }
    }

    void vector_to_cufftComplex(const std::vector<std::complex<double> > &x, hipfftComplex **complx, size_t Nx) 
    {
        (*complx) = (hipfftComplex*) malloc(sizeof(hipfftComplex) * Nx);

        for (int i  = 0; i < Nx; ++i)
        {
            (*complx)[i].x = x[i].real();
            (*complx)[i].y = x[i].imag();
        }
    }
}


int main() {
    int i;
    
    helpers::CPUTimer timer;

    std::vector<std::complex<double> > samples;
    dft_functions::generate_fake_signal(samples, N_DIM);

    hipfftHandle plan = 0;
    hipfftComplex *complexSamples, *complexSamples_d, *complexFreq;

    complexFreq = (hipfftComplex *) malloc( sizeof (hipfftComplex) * N_DIM);

    dft_functions::vector_to_cufftComplex(samples, &complexSamples, N_DIM);

  

    printf("Initial samples:\n");

    for(i=0; i < N_PRINTS; ++i) {
        printf(" %2.4f\n", samples[i]);
    }
    printf("...\n");

    timer.start();
    std::vector<std::complex<double>> complexFreq_h = dft_functions::dft(samples);
    double elapsed = timer.stop();
    printf("Naive impl. output samples:\n");

    for(i=0; i < N_PRINTS; ++i) {
        printf("  %d: (%2.4f, %2.4f)\n", i + 1, complexFreq_h[i].real(),
        complexFreq_h[i].imag());
    }
    printf("...\n");
    printf("Elapsed on CPU: %f \n", elapsed);

    // setup cuFFT plan
    CHECK_CUFFT(hipfftPlan1d(&plan, N_DIM, HIPFFT_C2C, 1));
    CHECK(hipMalloc((void**) &complexSamples_d, sizeof(hipfftComplex) * N_DIM));

    CHECK(hipMemcpy(complexSamples_d, complexSamples, sizeof(hipfftComplex) * N_DIM, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop);

    hipEventRecord(start);
    CHECK_CUFFT(hipfftExecC2C(plan, complexSamples_d, complexSamples_d, HIPFFT_FORWARD));
    hipEventRecord(stop);
    
    CHECK(hipMemcpy(complexFreq, complexSamples_d, sizeof(hipfftComplex) * N_DIM, hipMemcpyDeviceToHost));

    hipEventSynchronize(stop);
    float elapsed_d = 0;
    hipEventElapsedTime(&elapsed_d, start, stop);

    printf("Fourier coefficients: \n");
    for(i=0; i < N_PRINTS; ++i) {
        printf("  %d: (%2.4f, %2.4f)\n", i + 1, complexFreq[i].x,
               complexFreq[i].y);
    }
    printf("... \n");
    printf("Elapsed on Cuda: %f \n", elapsed_d);


    free(complexSamples);
    free(complexFreq);

    CHECK(hipFree(complexSamples_d));
    CHECK_CUFFT(hipfftDestroy(plan));

}